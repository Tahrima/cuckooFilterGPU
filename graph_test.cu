#include "hip/hip_runtime.h"
/*
 * Parallel Graph Preprocessing of cuckoo filter
 * This preprocesses a batch insertion into a cuckoo filter by creating a directed graph (V,E) where:
 *    V is a set of vertices that represent each bucket of the cuckoo filter
 *    E is a set of edges (u,v) with weight w where:
 *      w is the fingerprint of a specific entry
 *      u is the bucket number given by hash(entry)
 *      v is the bucket number given by hash(entry) xor hash(fingerprint)
 *      dir indicates the vertex pointed to by the edge. Also indicates
 *          which bucket number the fingerprint should be placed in.
 */

#include <cstring>
#include <stdexcept>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <climits>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hash/hash_functions.cu"

#define LARGE_THRESHOLD_VAL 10000
#define NUM_BUCKETS 5

__device__ uint64_t TwoIndependentMultiplyShift(uint64_t key) {
    const uint64_t SEED[4] = {0x818c3f78ull, 0x672f4a3aull, 0xabd04d69ull, 0x12b51f95ull};
    const uint64_t m = SEED[0];
    const uint64_t a = SEED[2];
    return (a + m * key) >> 56;
}

template <typename T_file>
void openFileToAccess( T_file& input_file, std::string file_name ) {
	input_file.open( file_name.c_str() );
	if( !input_file )
		throw std::runtime_error( "Failed to open specified file: " + file_name + "\n" );
}

void * cudaMallocAndCpy(int size, void * hostMemory) {
  void * gpuMem;
  hipMalloc((void**) &gpuMem, size);
  if (hostMemory != NULL) {
    hipMemcpy(gpuMem, hostMemory, size, hipMemcpyHostToDevice);
  }
  return gpuMem;
}

void cudaGetFromGPU(void * destination, void * gpuMemory, int size) {
  hipMemcpy(destination, gpuMemory, size, hipMemcpyDeviceToHost);
}

void cudaSendToGPU(void * destination, void * hostMemory, int size) {
  hipMemcpy(destination, hostMemory, size, hipMemcpyHostToDevice);
}

class Edge {
  public:
    unsigned int src; //hash(x) location
    unsigned int dst; //hash(x) xor hash(fp) location
    unsigned char fp; //fingerprint
    int dir; //0 to be src, 1 to be dst

 	__device__ __host__ Edge(){}
};

class Graph {
  public:
    int buckets[NUM_BUCKETS]; //value at index i is the number of indegrees to a bucket i
  	Edge *edges;
  	unsigned int num_edges;

    __device__ __host__ Graph(unsigned int max_bucket_size, unsigned int size) {
      num_edges = size;
      for(int i=0; i<NUM_BUCKETS; i++){
        buckets[i] = -max_bucket_size;
      }
      edges = NULL;
    }

    __device__ void printGraph() {
      int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
      if(thread_id == 0) {
        for(int i=0; i<num_edges; i++) {
          printf("Edge %u: %d \t src: %u \t dst: %u\n",i, edges[i].fp, edges[i].src, edges[i].dst);
        }
      }
    }
};

// __global__ void setup_kernel (hiprandState * state, Graph *g)
// {
//   	int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
//    	// change sequence number to currIdx if values are too correlated
//   	// hiprand_init(1234, 0, 0, &state[currIdx]);
//     hiprand_init(1234, 0, 0, &state[currIdx]);

// }


/**
 * Parallel graph building
 * @param entries is a list of entries to enter
 * @param entryListSize is the size of the @param entries list
 * @param g is an address in the GPU to place result. Assumes g->edges has been given enough space for @param entryListSize items
 */
__global__ void findAllCollisions(int* entries, int entryListSize, Graph * g) {
  int total_threads = blockDim.x * gridDim.x; //total threads
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
  int thread_id_block = threadIdx.x; //thread number in block


  // CHANGE BELOW LINE TO BE MORE EFFICIENT
  int rounds = entryListSize % total_threads == 0 ? (entryListSize/total_threads):((entryListSize/total_threads)+1);
  g->num_edges = entryListSize;

  for (size_t i = 0; i <rounds; i++) {
    int currIdx = rounds*total_threads + thread_id;
    int * entry = &entries[currIdx];

    unsigned int bucket1;
    hash_item((unsigned char*) entry,
                  4,
                  NUM_BUCKETS,
    		      HASHFUN_NORM,
                  &bucket1);

    const uint64_t hash = TwoIndependentMultiplyShift(*entry);

    unsigned char fp = (unsigned char) hash;
    unsigned int fpHash;
    hash_item((unsigned char*) &fp,
                  1,
                  NUM_BUCKETS,
    		      HASHFUN_NORM,
                  &fpHash);
    unsigned int bucket2 = (bucket1 ^ fpHash) & 0b11111111;

    //build edge
    g->edges[currIdx].fp = fp;
    g->edges[currIdx].src = bucket1 % NUM_BUCKETS;
    g->edges[currIdx].dst = bucket2 % NUM_BUCKETS;


// 	Copy state to local memory for efficiency */
//     hiprandState local_state = global_state[thread_id];
// 	/* Generate pseudo - random unsigned ints
//     g->edges[i].dir = hiprand_uniform(&local_state);

    //update bucket
    atomicAdd(&(g->buckets[bucket1]), 1);
  }
  syncthreads();
  g->printGraph();
}

/**
 * Edge Processing Kernel
 * Finds random edges to evict until capacity for each bucket is equal to 0
 *
 */
__global__ void processEdges(Graph * g, int* anyChange) {
  int total_threads = blockDim.x * gridDim.x; //total threads
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
  int thread_id_block = threadIdx.x; //thread number in block
  int num_edges = g->num_edges;

  int rounds = num_edges % total_threads == 0 ? (num_edges/total_threads):(num_edges/total_threads);

  for(int i=0; i<rounds; i++) {
  	int currIdx = total_threads*i + thread_id; //current edge to process
    Edge *e = &g->edges[currIdx];

    //determine the bucket it's in
    int curr_bucket = e->dir == 0 ? e->src:e->dst;

    //check the bucket
    int * bucketCount = &(g->buckets[curr_bucket]);
    int tmp = *bucketCount;

    //decrement the bucket count if > 0
    if(*bucketCount > 0) {
      printf("flipping direction\n");
      int old = atomicDec((unsigned int *)bucketCount, INT_MAX);
      if (old && old < LARGE_THRESHOLD_VAL) {
      	e->dir = e->dir ^ 1; // flip the bit
        *anyChange = 1;
      }
    }

  }
}

void initGraphCPU(int entry_size) {
	Graph * graph;
  	hipMalloc(&graph, sizeof(Graph));
    Edge * e;
    hipMalloc(&e, sizeof(Edge)*entry_size);
}

void insert(int* entries, unsigned int num_entries, unsigned int bucket_size, int num_buckets){
  std::cout << "Inserting " << num_entries << " entries"<< std::endl;
	int anychange = 1;
  	int * d_change = (int *) cudaMallocAndCpy(sizeof(int), &anychange);

  	Graph *h_graph = new Graph(bucket_size, num_entries);

  	//set up pointer
  	hipMalloc((void**)&(h_graph->edges), sizeof(Edge)*num_entries);
  	Graph *d_graph = (Graph *) cudaMallocAndCpy(sizeof(Graph), h_graph);
  	int * d_entries = (int *) cudaMallocAndCpy(sizeof(int)*num_entries, entries);

  	while (anychange != 0){
      std::cout << "Calling kernel" << std::endl;
      anychange = 0;
      cudaSendToGPU(d_change, &anychange, sizeof(int));

      findAllCollisions<<<2, 512>>>(d_entries, num_entries, d_graph);
      hipDeviceSynchronize();
      std::cout << "Found all collisions" << std::endl;

      processEdges<<<ceil(num_entries/1024), 1024>>>(d_graph, d_change);
      hipDeviceSynchronize();
      std::cout << "Proccessed edge" << std::endl;

      cudaGetFromGPU(&anychange, d_change, sizeof(int));
      std::cout << "Got value of anychange: " << anychange << std::endl;

    }
}
