//GPUBatchInserts.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#include "../mt19937ar.h"
#include "quotientFilter.cuh"

#ifndef NOT_FOUND
#define NOT_FOUND UINT_MAX
#endif

void generateRandomNumbers(unsigned int *numberArray, unsigned int n)
{
    init_genrand(time(NULL));   //initialize random number generator
    for (int i = 0; i < n; i++){
        numberArray[i] = genrand_int32();
    }
}

void CUDAErrorCheck()
{
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

int main(int argc, char* argv[])
{
    assert(argc == 5);
    int q = atoi(argv[1]);
    int r = atoi(argv[2]);
    float alpha = atof(argv[3]);    //initial fill %
    int batchSize = atoi(argv[4]);  //size of batch to insert after build

    //Initialize filter
    struct quotient_filter d_qfilter;
    initFilterGPU(&d_qfilter, q, r);
    hipMemset(d_qfilter.table, 0, calcNumSlotsGPU(q, r) * sizeof(unsigned char));
    
    //Generate set of random numbers
    int numValues = alpha * (1 << q);
    unsigned int* h_randomValues = new unsigned int[numValues];
    generateRandomNumbers(h_randomValues, numValues);
    unsigned int* d_randomValues;
    hipMalloc((void**) &d_randomValues, numValues * sizeof(unsigned int));
    hipMemcpy(d_randomValues, h_randomValues, numValues * sizeof(unsigned int), hipMemcpyHostToDevice);

//Random Inserts
    float filterBuildTime = insert(d_qfilter, numValues, d_randomValues, &d_qfilter.cardinality);
//    printf("Insert rate = %f million ops/sec\n", numValues / filterBuildTime / 1000);

//Insert new batch
    unsigned int* h_newBatch = new unsigned int[batchSize];
    generateRandomNumbers(h_newBatch, batchSize);
    unsigned int* d_newBatch;
    hipMalloc((void**) &d_newBatch, batchSize * sizeof(unsigned int));
    hipMemcpy(d_newBatch, h_newBatch, batchSize * sizeof(unsigned int), hipMemcpyHostToDevice);
    float insertTime = insert(d_qfilter, batchSize, d_newBatch, &d_qfilter.cardinality);
    printf("%f\n", batchSize / insertTime / 1000);

    //Free Memory
    hipFree(d_qfilter.table);
    delete[] h_randomValues;
    hipFree(d_randomValues);
    delete[] h_newBatch;
    hipFree(d_newBatch);
    hipDeviceReset();

    return 0;
}
