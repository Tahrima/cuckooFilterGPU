//GPUBatchInserts.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#include "../mt19937ar.h"
#include "quotientFilter.cuh"

#ifndef NOT_FOUND
#define NOT_FOUND UINT_MAX
#endif

void generateRandomNumbers(unsigned int *numberArray, unsigned int n)
{
    init_genrand(time(NULL));   //initialize random number generator
    for (int i = 0; i < n; i++){
        numberArray[i] = genrand_int32();
    }
}

void CUDAErrorCheck()
{
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

int main(int argc, char* argv[])
{
    assert(argc == 5);
    int q = atoi(argv[1]);
    int r = atoi(argv[2]);
    float alpha = atof(argv[3]);    //initial fill %
    int batchSize = atoi(argv[4]);  //size of batch to insert after build

    //TODO: Initialize filter
    /*struct quotient_filter d_qfilter;
    initFilterGPU(&d_qfilter, q, r);
    hipMemset(d_qfilter.table, 0, calcNumSlotsGPU(q, r) * sizeof(unsigned char));
    */
    //Generate set of random numbers
    int numValues = alpha * (1 << q);
    unsigned int* h_randomValues = new unsigned int[numValues];
    generateRandomNumbers(h_randomValues, numValues);
    unsigned int* d_randomValues;
    hipMalloc((void**) &d_randomValues, numValues * sizeof(unsigned int));
    hipMemcpy(d_randomValues, h_randomValues, numValues * sizeof(unsigned int), hipMemcpyHostToDevice);

//Random Inserts
    float filterBuildTime = insert(d_randomValues, numValues, d_randomValues, &d_qfilter.cardinality);
//    printf("Insert rate = %f million ops/sec\n", numValues / filterBuildTime / 1000);


//TODO: Insert new batch

    //Free Memory
    delete[] h_randomValues;
    hipFree(d_randomValues);
    hipDeviceReset();

    return 0;
}
