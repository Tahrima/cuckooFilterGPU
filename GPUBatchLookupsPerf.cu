#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#include "mt19937ar.h"
#include "CuckooFilter.cu"
#include "graph_test.cu"

#ifndef NOT_FOUND
#define NOT_FOUND UINT_MAX
#endif

void generateRandomNumbers(unsigned int *numberArray, unsigned int n)
{
    srand((unsigned int)time(NULL));
    for (int i = 0; i < n; i++){
        numberArray[i] = rand();
    }
}

void CUDAErrorCheck()
{
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

int main(int argc, char* argv[])
{

    assert(argc==4);
    unsigned int numBuckets = atoi(argv[1]);
    unsigned int bucketSize = atoi(argv[2]);
    float fillFraction = (float)atof(argv[3]);
    //Generate values for random lookups

    int insertSize = floor(numBuckets*bucketSize*fillFraction);
    unsigned int* h_insertValues = new unsigned int[insertSize];
    generateRandomNumbers(h_insertValues, insertSize);

    CuckooFilter * ckFilter = new CuckooFilter(numBuckets, bucketSize);
    insert((int *)h_insertValues, insertSize, numBuckets, bucketSize, ckFilter);

    // Lookup values are the inserted values from earlier.
    unsigned int * d_lookUpValues;
    hipMalloc((void**) &d_lookUpValues, insertSize * sizeof(unsigned int));
    hipMemcpy(d_lookUpValues, h_insertValues, insertSize * sizeof(unsigned int), hipMemcpyHostToDevice);

    //Output array
    char * d_results;
    hipMalloc((void**) &d_results, insertSize * sizeof(char));
    hipMemset(&d_results, 0, insertSize * sizeof(char));

    CuckooFilter * d_ckFilter = (CuckooFilter *) cudaMallocAndCpy(sizeof(CuckooFilter), ckFilter);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Launch lookup kernel
    hipProfilerStart();
    hipEventRecord(start);

    std::cout << "Calling lookup kernel" << std::endl;
    lookUpGPU<<<(insertSize + 1023)/1024, 1024>>>(d_ckFilter, insertSize, d_lookUpValues, d_results);
    hipDeviceSynchronize();
    char * h_results = new char[insertSize];
    hipMemcpy(h_results, d_results, insertSize* sizeof(char), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipProfilerStop();

    //Calculate and print timing results
    hipEventSynchronize(stop);
    float batchLookupTime = 0;
    hipEventElapsedTime(&batchLookupTime, start, stop);
    printf("%f\n", insertSize / batchLookupTime / 1000);
    //Free Memory
     delete[] h_insertValues;
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    hipFree(d_lookUpValues);
    hipFree(d_results);
    delete[] h_results;
    hipDeviceReset();

    return 0;
}
