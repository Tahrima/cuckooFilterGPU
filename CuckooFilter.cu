
#include <hip/hip_runtime.h>
class CuckooFilter {
  public:
    char** buckets;
    unsigned int numBuckets;
    unsigned int bucketSize;
    __host__ CuckooFilter(unsigned int numberOfBuckets, unsigned int bucketSizeIn) {
      numBuckets = numberOfBuckets;
      bucketSize = bucketSizeIn;



      char ** tmpbuckets = new char*[numberOfBuckets];
      for(int i=0; i<numBuckets; i++){
        hipMalloc((void**)&tmpbuckets[i], sizeof(char) * bucketSize);
      }
      hipMalloc((void**)&buckets, sizeof(char*)*numberOfBuckets);
      hipMemcpy(buckets, tmpbuckets, sizeof(char*)*numberOfBuckets, hipMemcpyHostToDevice);
    }
    __host__ void freeFilter() {
      for (int i = 0; i < numBuckets; i++) {
        hipFree(buckets[i]);
      }
      hipFree(buckets);
    }
    __device__ void insert(unsigned int fingerprint, unsigned int bucketNum, unsigned int index) {
      buckets[bucketNum][index] = fingerprint;
    }
    __device__ unsigned int lookup(unsigned int bucketNum, unsigned int index) {
      return(buckets[bucketNum][index]);
    }
    __device__ unsigned int lookupFingerprintInBucket(unsigned int fingerprint, unsigned int bucketNum) {
      int retVal = 0;
      for (int i = 0; i < bucketSize; i++) {
        retVal = retVal || (fingerprint == buckets[bucketNum][i]);
      }
      return(retVal);
    }

    __device__ void printFilter() {
      int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
      if(thread_id == 0) {
        for(int i=0; i<numBuckets; i++) {
          printf("Bucket %d: \t",i);
          for (int j = 0; j < bucketSize; j++) {
            printf(" | %d |", buckets[i][j]);
          }
          printf("\n");
        }
      }
    }
};

__global__ void lookupGPU(CuckooFilter *ck, unsigned int *fp, unsigned int* b1, unsigned int *b2, char * results, int num_fp){
    int total_threads = blockDim.x * gridDim.x; //total threads
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
    int rounds = (num_fp % total_threads == 0) ? (num_fp/total_threads):((num_fp/total_threads)+1);

    for (size_t i = 0; i < rounds; i++) {
      int currIdx = total_threads*i + thread_id;
      if(currIdx < num_fp){
        int in_b1 = ck->lookupFingerprintInBucket(fp[currIdx], b1[currIdx]);
        int in_b2 = ck->lookupFingerprintInBucket(fp[currIdx], b2[currIdx]);

        results[currIdx] = in_b1 || in_b2;

      }
    }
}
