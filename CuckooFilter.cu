
#include <hip/hip_runtime.h>
class CuckooFilter {
  public:
    char** buckets;
    unsigned int numBuckets;
    unsigned int bucketSize;
    __host__ CuckooFilter(unsigned int numberOfBuckets, unsigned int bucketSizeIn) {
      numBuckets = numberOfBuckets;
      bucketSize = bucketSizeIn;
      hipMalloc((void**)&buckets, sizeof(char*) * numBuckets);
      for(int i=0; i<numBuckets; i++){
        hipMalloc((void**)&buckets[i], sizeof(char) * bucketSize);
      }
    }
    __host__ void freeFilter() {
      for (int i = 0; i < numBuckets; i++) {
        hipFree(buckets[i]);
      }
      hipFree(buckets);
    }
    __device__ void insert(unsigned int fingerprint, unsigned int bucketNum, unsigned int index) {
      buckets[bucketNum][index] = fingerprint;
    }
    __device__ unsigned int lookup(unsigned int bucketNum, unsigned int index) {
      return(buckets[bucketNum][index]);
    }
    __device__ unsigned int lookupFingerprintInBucket(unsigned int fingerprint, unsigned int bucketNum) {
      int retVal = 0;
      for (int i = 0; i < bucketSize; i++) {
        retVal = retVal || (fingerprint == buckets[bucketNum][i]);
      }
      return(retVal);
    }

    __device__ void printFilter() {
      int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
      if(thread_id == 0) {
        for(int i=0; i<numBuckets; i++) {
          printf("Bucket %d: \t",i);
          for (int j = 0; j < bucketSize; j++) {
            printf(" | %d |", buckets[i][j]);
          }
          printf("\n");
        }
      }
    }
};
