#include "hip/hip_runtime.h"
#include <cstring>
#include <stdexcept>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <climits>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "hash/hash_functions.cu"

__device__ uint64_t TwoIndependentMultiplyShift(unsigned int key) {
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
    const uint64_t SEED[4] = {0x818c3f78ull, 0x672f4a3aull, 0xabd04d69ull, 0x12b51f95ull};
    const uint64_t m = SEED[0];
    const uint64_t a = SEED[2];
    //printf("thread: %d \t key: %u, m: %u, a: %u = %lu\n",thread_id, key, m, a, (a + m * key));
    return (a + m * key);
}

class CuckooFilter {
  public:
    char** buckets;
    unsigned int numBuckets;
    unsigned int bucketSize;
    __host__ CuckooFilter(unsigned int numberOfBuckets, unsigned int bucketSizeIn) {
      numBuckets = numberOfBuckets;
      bucketSize = bucketSizeIn;



      char ** tmpbuckets = new char*[numberOfBuckets];
      for(int i=0; i<numBuckets; i++){
        hipMalloc((void**)&tmpbuckets[i], sizeof(char) * bucketSize);
        hipMemset((tmpbuckets[i]), 0, sizeof(char) * bucketSize);
      }
      hipMalloc((void**)&buckets, sizeof(char*)*numberOfBuckets);
      hipMemcpy(buckets, tmpbuckets, sizeof(char*)*numberOfBuckets, hipMemcpyHostToDevice);
    }
    __host__ void freeFilter() {
      char ** tmpBuckets = new char*[bucketSize];
      hipMemcpy(tmpBuckets, tmpBuckets, sizeof(char*)*numBuckets, hipMemcpyDeviceToHost);
      for (int i = 0; i < numBuckets; i++) {
        hipFree(tmpBuckets[i]);
      }
      hipFree(buckets);
    }
    __device__ void insert(char fingerprint, unsigned int bucketNum, unsigned int index) {
      buckets[bucketNum][index] = fingerprint;
    }
    __device__ char lookup(unsigned int bucketNum, unsigned int index) {
      return(buckets[bucketNum][index]);
    }
    __device__ char lookupFingerprintInBucket(char fingerprint, unsigned int bucketNum) {
      char retVal = 0;
      for (int i = 0; i < bucketSize; i++) {
        retVal = retVal || (fingerprint == buckets[bucketNum][i]);
      }
      return(retVal);
    }

    __device__ void printFilter() {
      int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
      if(thread_id == 0) {
        for(int i=0; i<numBuckets; i++) {
          printf("Bucket %d: \t",i);
          for (int j = 0; j < bucketSize; j++) {
            printf(" | %u |", (unsigned char)buckets[i][j]);
          }
          printf("\n");
        }
      }
    }
    __device__ void printBucket(unsigned int numBucket) {
      int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
        for(int i=numBucket; i<=numBucket; i++) {
          printf("Thread %d, Bucket %d: \t", thread_id, i);
          for (int j = 0; j < bucketSize; j++) {
            printf(" | %u |", (unsigned char)buckets[i][j]);
          }
          printf("\n");
        }
    }
};

__global__ void lookUpGPU(CuckooFilter *ck, int numLookUps, unsigned int *lookUps, char * results){

    int total_threads = blockDim.x * gridDim.x; //total threads
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x; //real thread number
    int rounds = (numLookUps % total_threads == 0) ? (numLookUps/total_threads):((numLookUps/total_threads)+1);

    for (size_t i = 0; i < rounds; i++) {
      int currIdx = total_threads*i + thread_id;
      if(currIdx < numLookUps){

        unsigned int entry = lookUps[currIdx];
        unsigned int bucket1;
        hash_item((unsigned char*) &entry,
                      4,
                      ck->numBuckets,
                      HASHFUN_NORM,
                      &bucket1);

        const uint64_t hash = TwoIndependentMultiplyShift(entry);
        unsigned char fp = (unsigned char) hash;
        unsigned int fpHash;
        hash_item((unsigned char*) &fp,
                      1,
                      ck->numBuckets,
                      HASHFUN_NORM,
                      &fpHash);
        unsigned int bucket2 = ((bucket1 ^ fpHash) & 0b11111111) % ck->numBuckets;

        int in_b1 = ck->lookupFingerprintInBucket(fp, bucket1);
        int in_b2 = ck->lookupFingerprintInBucket(fp, bucket2);

        results[currIdx] = (char) (in_b1 || in_b2);
        // if (!results[currIdx]){
        //     printf("Entry = %u, fp = %u, b%u=%u, b%u=%u, results=%d, ACTUAL %u\n", entry, (unsigned char)fp, bucket1, in_b1, bucket2, in_b2, in_b1 || in_b2, results[currIdx]);
        //     ck->printBucket(bucket1);
        //     ck->printBucket(bucket2);
        // }
      }
    }
    __syncthreads();
}
