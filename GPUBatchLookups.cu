#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#include "../mt19937ar.h"

#ifndef NOT_FOUND
#define NOT_FOUND UINT_MAX
#endif

void generateRandomNumbers(unsigned int *numberArray, unsigned int n)
{
    init_genrand(time(NULL));   //initialize random number generator
    for (int i = 0; i < n; i++){
        numberArray[i] = genrand_int32();
    }
}

void CUDAErrorCheck()
{
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

int main(int argc, char* argv[])
{

    unsigned int numBuckets = atoi(argv[1]);
    unsigned int bucketSize = atoi(argv[2]);
    unsigned int numValues = atoi(argv[3]);
    //New random batch lookups
    //Generate values for random lookups
    unsigned int* h_batchLookupValues = new unsigned int[batchSize];
    generateRandomNumbers(h_batchLookupValues, batchSize);

    //Array of lookup values
    unsigned int* d_batchLookupValues;
    hipMalloc((void**) &d_batchLookupValues, batchSize * sizeof(int));
    hipMemcpy(d_batchLookupValues, h_batchLookupValues, batchSize * sizeof(int), hipMemcpyHostToDevice);

    //Output array
    unsigned int* d_batchReturnValues;
    hipMalloc((void**) &d_batchReturnValues, batchSize * sizeof(unsigned int));
    hipMemset(&d_batchReturnValues, 0, batchSize * sizeof(unsigned int));


    CuckooFilter ckFilter = new CuckooFilter(numBuckets, bucketSize);

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    //Launch lookup kernel
    // hipProfilerStart();
    // hipEventRecord(start);
    lookUp<<<(batchSize + 1023)/1024, 1024>>>(batchSize, d_qfilter, d_batchLookupValues, d_batchReturnValues);
    // hipEventRecord(stop);
    // hipProfilerStop();

    //Calculate and print timing results
    // hipEventSynchronize(stop);
    // float batchLookupTime = 0;
    // hipEventElapsedTime(&batchLookupTime, start, stop);
//    printf("Random lookup rate = %f million ops/sec\n", numValues / randomLookupTime / 1000);
    //printf("%f\n", batchSize / batchLookupTime / 1000);

    //Free Memory
    hipFree(d_qfilter.table);
    delete[] h_randomValues;
    hipFree(d_randomValues);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    delete[] h_batchLookupValues;
    hipFree(d_batchLookupValues);
    hipFree(d_batchReturnValues);
    hipDeviceReset();

    return 0;
}
