#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#include "mt19937ar.h"
#include "CuckooFilter.cu"
#include "graph_test.cu"

#ifndef NOT_FOUND
#define NOT_FOUND UINT_MAX
#endif

void generateRandomNumbers(unsigned int *numberArray, unsigned int n)
{
    init_genrand(time(NULL));   //initialize random number generator
    for (int i = 0; i < n; i++){
        numberArray[i] = genrand_int32();
    }
}

void CUDAErrorCheck()
{
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

int main(int argc, char* argv[])
{

    unsigned int numBuckets = atoi(argv[1]);
    unsigned int bucketSize = atoi(argv[2]);
    float fillFraction = (float)atof(argv[3]);
    unsigned int numLookUps = atoi(argv[4]);
    //New random batch lookups
    //Generate values for random lookups

    int insertSize = floor(numBuckets*bucketSize*fillFraction);
    unsigned int* h_insertValues = new unsigned int[insertSize];
    generateRandomNumbers(h_insertValues, insertSize);

    CuckooFilter * ckFilter = new CuckooFilter(numBuckets, bucketSize);
    insert((int *)h_insertValues, insertSize, numBuckets, bucketSize, ckFilter);

    unsigned int* h_lookUpValues = new unsigned int[numLookUps];
    generateRandomNumbers(h_insertValues, numLookUps);

    unsigned int * d_lookUpValues;
    hipMalloc((void**) &d_lookUpValues, numLookUps * sizeof(unsigned int));
    hipMemcpy(&d_lookUpValues, &h_lookUpValues, numLookUps * sizeof(unsigned int), hipMemcpyHostToDevice);

    //Output array
    char * d_results;
    hipMalloc((void**) &d_results, numLookUps * sizeof(char));
    hipMemset(&d_results, 0, numLookUps * sizeof(char));
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    //Launch lookup kernel
    // hipProfilerStart();
    // hipEventRecord(start);
    lookUpGPU<<<(numLookUps + 1023)/1024, 1024>>>(ckFilter, numLookUps, d_lookUpValues, d_results);
    hipDeviceSynchronize();
    char * h_results = new char[numLookUps];
    hipMemcpy(&h_results, &d_results, numLookUps* sizeof(char), hipMemcpyDeviceToHost);
    // hipEventRecord(stop);
    // hipProfilerStop();

    //Calculate and print timing results
    // hipEventSynchronize(stop);
    // float batchLookupTime = 0;
    // hipEventElapsedTime(&batchLookupTime, start, stop);
//    printf("Random lookup rate = %f million ops/sec\n", numValues / randomLookupTime / 1000);
    //printf("%f\n", batchSize / batchLookupTime / 1000);

    //Free Memory
    ckFilter->freeFilter();
    delete[] h_insertValues;
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    delete[] h_lookUpValues;
    hipFree(d_lookUpValues);
    hipFree(d_results);
    delete[] h_results;
    hipDeviceReset();

    return 0;
}
